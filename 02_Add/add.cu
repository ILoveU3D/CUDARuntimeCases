
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 20*65536

void add(int *a,int *b,int *c){
	int t=0;
	while(t<N){
		c[t]=a[t]+b[t];
		t++;
	}
}

int main(){
	int a[N],b[N],c[N];
	//赋值
	for(int i=0;i<N;i++){
		a[i]=i-3;
		b[i]=i/2+1;
	}
	time_t start,end;
	start = time(NULL);
	add(a,b,c);
	end = time(NULL);
	printf("time=%fs\n",difftime(end,start));
}
