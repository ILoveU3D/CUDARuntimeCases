
#include <hip/hip_runtime.h>
#include<stdio.h>
#define DIMX 10
#define DIMY 10

__global__ void add(int *a,int *b,int *c){
	int x=blockIdx.x;
	int y=blockIdx.y;
	int offset=x*gridDim.y+y;
	if(x<DIMX&&y<DIMY)
		c[offset]=a[offset]+b[offset];
}

int main(){
	int a[DIMX*DIMY],b[DIMX*DIMY],c[DIMX*DIMY];
	int *a_cuda,*b_cuda,*c_cuda;
	//赋值
	for(int i=0;i<DIMX;i++){
		for(int j=0;j<DIMY;j++){
			a[i*DIMX+j]=i+j-3;
			b[i*DIMX+j]=(i+j)/2+1;
		}
	}
	hipMalloc((void**)&a_cuda,DIMX*DIMY*sizeof(int));
	hipMalloc((void**)&b_cuda,DIMX*DIMY*sizeof(int));
	hipMalloc((void**)&c_cuda,DIMX*DIMY*sizeof(int));
	hipMemcpy(a_cuda,a,DIMX*DIMY*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(b_cuda,b,DIMX*DIMY*sizeof(int),hipMemcpyHostToDevice);
	dim3 grid(DIMX,DIMY);
	add<<<grid,1>>>(a_cuda,b_cuda,c_cuda);
	hipMemcpy(c,c_cuda,DIMX*DIMY*sizeof(int),hipMemcpyDeviceToHost);
	printf("a+b=(");
	for(int i=0;i<DIMX;i++){
		for(int j=0;j<DIMY;j++){
			printf("%d,",c[i*DIMX+j]);
		}
		printf("\n");
	}
	printf(")\n");
	hipFree(a_cuda);
	hipFree(b_cuda);
	hipFree(c_cuda);
}
