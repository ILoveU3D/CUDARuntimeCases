#include "hip/hip_runtime.h"
#include<opencv2/opencv.hpp>
#include<math.h>
#include<stdio.h>
#define W 512
#define H 512
#define THREADS 32
using namespace cv;

__device__ float normalize(float *u,float *d, int offset){
	return d[offset]!=0.0 ? u[offset]/d[offset] : 0;
}

__global__ void cal_uxy(float *u,float *u_x, float *u_y){
	int x = threadIdx.x+blockIdx.x*blockDim.x;
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	if(x+1<W)
		u_x[x*W+y] = u[(x+1)*W+y] - u[x*W+y];
	else
		u_x[x*W+y] = u[x*W+y];
	if(y+1<H)
		u_y[x*W+y] = u[x*W+y+1] - u[x*W+y];
	else
		u_y[x*W+y] = u[x*W+y];
}

__global__ void cal_ud(float *u_x,float *u_y,float *u_d){
	int x = threadIdx.x+blockIdx.x*blockDim.x;
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	u_d[x*W+y] = sqrt(u_x[x*W+y]*u_x[x*W+y]+u_y[x*W+y]*u_y[x*W+y]);
}

__global__ void cal_uxy1(float *u_x,float *u_y,float *u_d,float *u_x1,float *u_y1){
	int x = threadIdx.x+blockIdx.x*blockDim.x;
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	if(x-1>=0)
		u_x1[x*W+y] = normalize(u_x,u_d,x*W+y) - normalize(u_x,u_d,(x-1)*W+y);
	else
		u_x1[x*W+y] = normalize(u_x,u_d,x*W+y);
	if(y-1>=0)
		u_y1[x*W+y] = normalize(u_y,u_d,x*W+y) - normalize(u_y,u_d,x*W+y-1);
	else
		u_y1[x*W+y] = normalize(u_y,u_d,x*W+y);
}

__global__ void cal_new(float *u,float *u0,float *u_x1,float *u_y1,float *u_temp, const float *lambda, const float *t){
	int x = threadIdx.x+blockIdx.x*blockDim.x;
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	float grad = *lambda * (u[x*W+y]-u0[x*W+y]) - u_x1[x*W+y] - u_y1[x*W+y];
	u_temp[x*W+y] = u[x*W+y] - *t * grad;
}

int main(){
	Mat img = imread("./tv/demo.jpeg",IMREAD_GRAYSCALE);
	img.convertTo(img, CV_32F);
	Mat result(W,H,CV_32F);
	const float t = 1.0, lambda = 0.01;
	int epoch = 20;
	float *u0,*u,*u_temp;
	float *u_x,*u_y,*u_d,*u_x1,*u_y1,*lambda_cuda,*t_cuda;
	hipMalloc((void**)&u0,W*H*sizeof(float));
	hipMalloc((void**)&u_x,W*H*sizeof(float));
	hipMalloc((void**)&u_y,W*H*sizeof(float));
	hipMalloc((void**)&u,W*H*sizeof(float));
	hipMalloc((void**)&u_temp,W*H*sizeof(float));
	hipMalloc((void**)&u_d,W*H*sizeof(float));
	hipMalloc((void**)&u_x1,W*H*sizeof(float));
	hipMalloc((void**)&u_y1,W*H*sizeof(float));
	hipMalloc((void**)&lambda_cuda,sizeof(float));
	hipMalloc((void**)&t_cuda,sizeof(float));
	hipMemcpy(u0, (float*)img.data, W*H*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(u, (float*)img.data, W*H*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(t_cuda, &t, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(lambda_cuda, &lambda, sizeof(float), hipMemcpyHostToDevice);
	dim3 GRID(W/THREADS,H/THREADS);
	dim3 BLOCK(THREADS,THREADS);
	for(int i=0;i<epoch;i++){
		printf("iterate %d ...\n",i);
		cal_uxy<<<GRID,BLOCK>>>(u,u_x,u_y);
		cal_ud<<<GRID,BLOCK>>>(u_x,u_y,u_d);
		cal_uxy1<<<GRID,BLOCK>>>(u_x,u_y,u_d,u_x1,u_y1);
		cal_new<<<GRID,BLOCK>>>(u,u0,u_x1,u_y1,u_temp,lambda_cuda,t_cuda);
		hipMemcpy(u, u_temp, W*H*sizeof(float),hipMemcpyDeviceToDevice);
	}
	hipMemcpy((float*)result.data, u, W*H*sizeof(float),hipMemcpyDeviceToHost);
	imwrite("./tv/result.bmp",result);
	hipFree(u0);
	hipFree(u);
	hipFree(u_temp);
	hipFree(u_x);
	hipFree(u_y);
	hipFree(u_d);
	hipFree(u_x1);
	hipFree(u_y1);
	hipFree(lambda_cuda);
	hipFree(t_cuda);
	return 0;
}
