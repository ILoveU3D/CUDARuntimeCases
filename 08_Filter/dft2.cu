#include "hip/hip_runtime.h"
#include<hipfft/hipfft.h>
#include<cmath>
#include<opencv2/opencv.hpp>
#define N 256
#define THREAD 16
using namespace cv;

__global__ void lowPass(hipfftComplex *data, float d0){
	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;
	uint offset = x*N + y;
	float dx = fabs((float)x - N/2);
	float dy = fabs((float)y - N/2);
	float d = sqrt(dx*dx + dy*dy);
	float filter;
	//ILPF
	//filter = d<=d0?1:0;
	//GLPF
	//filter = exp(-d*d/2/d0/d0);
	//BLPF
	filter = 1/(1+(d/d0)*(d/d0));
	data[offset].x *= filter;
	data[offset].y *= filter;
}

int main(){
	Mat img = imread("fft/test.jpg", IMREAD_GRAYSCALE);
	img.convertTo(img, CV_32F);
	Mat dft(N,N,CV_32F);
	hipfftComplex* data;
	hipfftHandle plan;
	hipMallocManaged((void **)&data, N*N*sizeof(hipfftComplex));
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			float shift = (i+j)%2==0?1:-1;
			data[i*N+j].x = img.at<float>(i,j) * shift;
		}
	}
	hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	dim3 grid(N/THREAD,N/THREAD);
	dim3 block(THREAD,THREAD);
	lowPass<<<grid,block>>>(data, 20);
	hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			float shift = (i+j)%2==0?1:-1;
			dft.at<float>(i,j) = data[i*N+j].x / (N*N) * shift;
		}
	}
	imwrite("fft/dft.jpg", dft);
	hipFree(data);
	hipfftDestroy(plan);
	return 0;
}
