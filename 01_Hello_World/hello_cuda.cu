
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernel(int a, int b, int *c){
	*c = a + b;
}

int main(){
	int c = 20;
	int *c_cuda;
	hipMalloc((void**)&c_cuda,sizeof(int));
	kernel<<<1,1>>>(1,1,c_cuda);
	hipMemcpy(&c,c_cuda,sizeof(int),hipMemcpyDeviceToHost);
	printf("c=%d\n",c);
	hipFree(c_cuda);
	return 0;
}
