#include "hip/hip_runtime.h"
#include<stdio.h>
#include<opencv2/opencv.hpp>
#include<math.h>
#define WIDTH 1024
#define HEIGHT 1024
#define FRAMES 108
using namespace cv;

__global__ void generate(uchar *data){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int t = blockIdx.z;
	int offset = x + y*blockDim.x*gridDim.x + t*blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	float fx = (float)x - gridDim.x*blockDim.x/2;
	float fy = (float)y - gridDim.y*blockDim.y/2;
	float d = sqrt(fx*fx+fy*fy);
	float mag = 10.0f;
	float value = 128.0f + 127.0f*cos((d-t)/mag)/(d/mag+1.0f);
	data[offset] = (uchar)value;
}

int main(){
	uchar *data_cuda;
	Mat img(WIDTH, HEIGHT, CV_8U);
	char name[10];
	hipMalloc((void**)&data_cuda,FRAMES*WIDTH*HEIGHT*sizeof(uchar));
	int threadPerBlock = 32;
	dim3 grid(WIDTH/threadPerBlock,HEIGHT/threadPerBlock,FRAMES);
	dim3 block(threadPerBlock,threadPerBlock);
	generate<<<grid,block>>>(data_cuda);
	for(int i=0;i<FRAMES;i++){
		uchar* t = data_cuda+WIDTH*HEIGHT*i;
		hipMemcpy(img.data,t,WIDTH*HEIGHT*sizeof(uchar),hipMemcpyDeviceToHost);
		sprintf(name,"./frames/%d.bmp",i);
		imwrite(name,img);
	}
	return 0;
}
