#include<cstdio>
#include<vector>
#include<hip/hip_runtime.h>
#include<hipblas.h>
using namespace std;

int main(){
    /* --- step 1. create handle --- */
	hipblasHandle_t handle = NULL;
	hipblasCreate(&handle);

    /* --- step 2. copy data to GPU --- */
	vector<float> a = {1,1,0,1};
	vector<float> b = {1,2,3,4};
	vector<float> c(4);
	const float alpha = 1;
	const float beta = 0;
	float *a_dev = nullptr;
	float *b_dev = nullptr;
	float *c_dev = nullptr;
	hipMalloc((void **)&a_dev, a.size() * sizeof(float));
	hipMalloc((void **)&b_dev, b.size() * sizeof(float));
	hipMalloc((void **)&c_dev, c.size() * sizeof(float));
	hipMemcpy(a_dev, a.data(), a.size()*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b.data(), b.size()*sizeof(float), hipMemcpyHostToDevice);

    /* --- step 3. compute by cuBLAS functions --- */
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 2, 2, 2, &alpha, a_dev, 2, b_dev, 2, &beta, c_dev, 2);
	
    /* --- step 4. return data to CPU and show ---*/
	hipMemcpy(c.data(), c_dev, c.size()*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0;i<c.size();i++)
		printf("%f ",c[i]);
	printf("\n");

    /* --- step 5.free memory ---*/
	hipFree(a_dev);
	hipFree(b_dev);
	hipFree(c_dev);
	hipblasDestroy(handle);
	return 0;
}
