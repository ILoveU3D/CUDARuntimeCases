#include "hip/hip_runtime.h"
#include<opencv2/opencv.hpp>
#include<stdio.h>
#define N 512
#define THREADS 32
#define LIFE 0
#define DEATH 255
using namespace cv;

texture<uchar, 2> map;

__device__ inline bool alive(uchar pixel){
	return pixel > uchar(128)? 0:1;
}

__global__ void evolute(uchar *after){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	uchar value = tex2D(map,x,y);
	int sum = 0;
	bool north = alive(tex2D(map,x,y-1));
	sum += north;
	bool south = alive(tex2D(map,x,y+1));
	sum += south;
	bool east = alive(tex2D(map,x+1,y));
	sum += east;
	bool west = alive(tex2D(map,x-1,y));
	sum += west;
	bool northeast = alive(tex2D(map,x+1,y-1));
	sum += northeast;
	bool northwest = alive(tex2D(map,x-1,y-1));
	sum += northwest;
	bool southeast = alive(tex2D(map,x+1,y+1));
	sum += southeast;
	bool southwest = alive(tex2D(map,x-1,y+1));
	sum += southwest;
	if(alive(value)){
		if(sum<2||sum>5)
			value = DEATH;
		else if(north&&south&&east&&west)
			value = DEATH;
		else
			value = LIFE;
	}else{
		if(sum==4)
			value = LIFE;
		else if(north&&south||east&&west)
			value = LIFE;
		else
			value = DEATH;
	}
	after[x*N+y] = value;
}

int main(){
	Mat img = imread("init.png",IMREAD_GRAYSCALE);
	uchar *map_cuda,*t;
	int times = 100;
	char name[20];
	hipMalloc((void**)&map_cuda, N*N*sizeof(uchar));
	hipMalloc((void**)&t, N*N*sizeof(uchar));
	hipMemcpy(map_cuda, img.data, N*N*sizeof(uchar), hipMemcpyHostToDevice);
	hipChannelFormatDesc desc = hipCreateChannelDesc<uchar>();
	hipBindTexture2D(NULL, map, map_cuda, desc, N, N, sizeof(uchar)*N);
	dim3 GRID(N/THREADS,N/THREADS);
	dim3 BLOCK(THREADS,THREADS);
	for(int i=0;i<times;i++){
		evolute<<<GRID,BLOCK>>>(t);
		hipMemcpy(img.data, map_cuda, N*N*sizeof(uchar), hipMemcpyDeviceToHost);
		sprintf(name, "./lifegame/%d.png", i);
		imwrite(name, img);
		hipMemcpy(map_cuda,t,N*N*sizeof(uchar),hipMemcpyDeviceToDevice);
	}
	hipUnbindTexture(map);
	hipFree(map_cuda);
	hipFree(t);
	return 0;
}
