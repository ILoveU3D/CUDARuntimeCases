
#include <hip/hip_runtime.h>
#include<stdio.h>
const int N = 100;
const int BLOCK = 128;

__device__ int sum(int *cache, int id){
	int i = blockDim.x/2;
	while(i!=0){
		if(id < i){
			cache[id] += cache[id+i];
		}
		__syncthreads();
		i /= 2;
	}
	return cache[0];
}

__global__ void dot(int *a,int *b,int *c){
	__shared__ int cache[BLOCK];
	int t = threadIdx.x;
	if(t < N)
		cache[t] = a[t]*b[t];
	else
		cache[t] = 0;
	__syncthreads();
	sum(cache,t);
	__syncthreads();
	*c = cache[0];
}

__host__ int dot(int *a,int *b){
	int *a_cuda,*b_cuda,*c_cuda;
	int r;
	hipMalloc((void**)&a_cuda,N*sizeof(int));
    hipMalloc((void**)&b_cuda,N*sizeof(int));
    hipMalloc((void**)&c_cuda,sizeof(int));
    hipMemcpy(a_cuda,a,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(b_cuda,b,N*sizeof(int),hipMemcpyHostToDevice);
	dot<<<1,BLOCK>>>(a_cuda,b_cuda,c_cuda);
	hipMemcpy(&r,c_cuda,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(a_cuda);
	hipFree(b_cuda);
	hipFree(c_cuda);
	return r;
}

int main(){
    int a[N],b[N],c;
    for(int i=0;i<N;i++){
    	a[i] = i;
    	b[i] = 1;
    }
    c = dot(a,b);
	printf("dot(a,b)=%d\n",c);
	return 0;
}
