#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<hiprand/hiprand_kernel.h>
#define THREADS 16
#define N 65536
#define XMAX 3.14
#define XMIN 0
#define YMAX 3.14
#define YMIN 0

__device__ float function(float x, float y){
	return y*sin(x);
}

__global__ void monte_carlo(float* mark, hiprandState* state, unsigned long seed){
	uint t = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, t, 0, &state[t]);
	float x = hiprand_uniform(&state[t]) * (XMAX-XMIN) + XMIN;
	float y = hiprand_uniform(&state[t]) * (YMAX-YMIN) + YMIN;
	float f = function(x,y);
	atomicAdd(mark, f * (XMAX-XMIN) * (YMAX-YMIN));
}

int main(){
	hiprandState* state;
	float *mark;
	hipMalloc(&state, N*sizeof(hiprandState));
	hipMallocManaged((void **)&mark, sizeof(float));
	*mark = 0;
	monte_carlo<<<N/THREADS,THREADS>>>(mark, state, time(NULL));
	hipDeviceSynchronize();
	printf("int is:%f\n", *mark/N);
	hipFree(mark);
	hipFree(state);
	return 0;
}