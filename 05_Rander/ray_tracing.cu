#include "hip/hip_runtime.h"
#include<stdio.h>
#include<opencv2/opencv.hpp>
#include"hip/hip_vector_types.h"
#include"Sphere.cu"
#define SPHERES 20
#define THREADS 32
#define FRAMES 108
using namespace cv;

__constant__ Sphere s_cuda[SPHERES*sizeof(Sphere)];

__global__ void ray_tracing(uchar *img){
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int t = blockIdx.z;
	int offset = x + y*blockDim.x*gridDim.x + t*blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	float fx = (float)x - gridDim.x*blockDim.x/2;
	float fy = (float)y - gridDim.y*blockDim.y/2;
	float2 origin = make_float2(fx,fy);
	float3 color = make_float3(0,0,0);
	float max_z = -INF;
	float3 c;
	for(int i=0;i<SPHERES;i++){
		float z = s_cuda[i].hit(origin,&c,t);
		if(z>max_z){
			color = c;
			max_z = z;
		}
	}
	img[offset*3+0] = (uchar)color.z;
	img[offset*3+1] = (uchar)color.y;
	img[offset*3+2] = (uchar)color.x;
}

int main(){
	Sphere s[SPHERES];
	Mat img(DIM,DIM,CV_8UC3);
	uchar *img_cuda;
	char name[10];
	for(int i=0;i<SPHERES;i++)
		s[i].init();
	hipMalloc((void**)&img_cuda,DIM*DIM*3*FRAMES*sizeof(uchar));
	hipMemcpyToSymbol(HIP_SYMBOL(s_cuda),s,sizeof(Sphere)*SPHERES);
	dim3 GRID(DIM/THREADS,DIM/THREADS,FRAMES);
	dim3 BLOCK(THREADS,THREADS);
	ray_tracing<<<GRID,BLOCK>>>(img_cuda);
	for(int i=0;i<FRAMES;i++){
		uchar* t = img_cuda+DIM*DIM*3*i;
		hipMemcpy(img.data,t,DIM*DIM*3*sizeof(uchar),hipMemcpyDeviceToHost);
		sprintf(name,"./tracing/%d.bmp",i);
		imwrite(name,img);
	}
	hipFree(img_cuda);
	return 0;
}
