
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 10*65536

__global__ void add(int *a,int *b,int *c){
	int t=blockIdx.x*blockDim.x+threadIdx.x;
	if(t<N)
		c[t]=a[t]+b[t];
}

int main(){
	int a[N],b[N],c[N];
	int *a_cuda,*b_cuda,*c_cuda;
	//赋值
	for(int i=0;i<N;i++){
		a[i]=i-3;
		b[i]=i/2+1;
	}
	time_t start,end;
	start = time(NULL);
	hipMalloc((void**)&a_cuda,N*sizeof(int));
	hipMalloc((void**)&b_cuda,N*sizeof(int));
	hipMalloc((void**)&c_cuda,N*sizeof(int));
	hipMemcpy(a_cuda,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(b_cuda,b,N*sizeof(int),hipMemcpyHostToDevice);
	add<<<N/128,128>>>(a_cuda,b_cuda,c_cuda);
	hipMemcpy(c,c_cuda,N*sizeof(int),hipMemcpyDeviceToHost);
	end = time(NULL);
	printf("time=%fs\n",difftime(end,start));
	hipFree(a_cuda);
	hipFree(b_cuda);
	hipFree(c_cuda);
}
