#include "hip/hip_runtime.h"
#include"hip/hip_vector_types.h"
#define DIM 512
#define MAX_SPEED 10
#define INF 2e10f
#define RAND(x) ((float)x*rand()/RAND_MAX)

struct Sphere{
	float3 position;
	float3 color;
	float3 velocity;
	float radius;
	__host__ void init(){
		position = make_float3(RAND(DIM)-DIM/2.0,RAND(DIM)-DIM/2.0,RAND(DIM/2)-DIM/4.0);
		color = make_float3(RAND(1),RAND(1),RAND(1));
		velocity = make_float3(RAND(MAX_SPEED)-MAX_SPEED/2.0,RAND(MAX_SPEED)-MAX_SPEED/2.0,0);
		radius = 25;
	}
	__device__ float hit(float2 camera,float3 *r,int time){
		float3 position_current = position + velocity*time;
		float distance = length(make_float2(camera.x-position_current.x,camera.y-position_current.y));
		if(distance < radius){
			float dz = sqrt(radius*radius - distance*distance);
			*r = color * dz / radius * 255;
			return position_current.z + dz; 
		}
		return -INF;
	}
};
